
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <arpa/inet.h>

#define BUFFER_LEN         64 
#define BUFFER_SIZE_OFFSET 56
//#define THREADS            4096 
#define THREADS            64 
#define LEFTROTATE(x, c)  (((x) << (c)) | ((x) >> (32 - (c))))


size_t pad(const char * message, uint32_t buffer[]) {
   size_t buffer_size = strnlen(message,BUFFER_LEN);
   uint8_t * buffer_8 = (uint8_t *) buffer;
   memset(buffer_8,0,BUFFER_LEN );
   memcpy(buffer_8,message,buffer_size);
   buffer_8[buffer_size] = 0x80;
   size_t newLen = buffer_size * 8;
   uint8_t * offset = &buffer_8[BUFFER_SIZE_OFFSET];
   memcpy(offset, &newLen, 4);
   return buffer_size;
}

void loadHash(char* argv, uint32_t* a, uint32_t* b, uint32_t* c, uint32_t* d) {
   char tmp[8];

   strncpy(tmp,&argv[0],8);
   *a = htonl(strtoul(tmp,0,16));

   strncpy(tmp,&argv[8],8);
   *b = htonl(strtoul(tmp,0,16));

   strncpy(tmp,&argv[16],8);
   *c = htonl(strtoul(tmp,0,16));

   strncpy(tmp,&argv[24],8);
   *d = htonl(strtoul(tmp,0,16));
}

__global__ void find( uint32_t hostA, uint32_t hostB, uint32_t hostC, uint32_t hostD, uint32_t * outM, char * msg ) {
    
   int tid = blockIdx.x;
   char init = ';' + (tid / 64);
   char stop = ';' + (tid % 64);

   //printf("I am thread %d and I will explore from %c%c;;;; to %c%czzzz a hash \n", tid, init, stop, init,stop);

   uint32_t targetA = hostA;
   uint32_t targetB = hostB;
   uint32_t targetC = hostC;
   uint32_t targetD = hostD;

   uint32_t M[BUFFER_LEN];
   for (int i=0; i<BUFFER_LEN; ++i) {
     M[i] = outM[i];
   } 


   char char0,char1,char2,char3,char4; 

   char0 = stop;
   for (char1 = ';'; char1 <= 'z' ; ++char1) {
 
   for (char2 = ';'; char2 <= 'z' ; ++char2) {
   for (char3 = ';'; char3 <= 'z' ; ++char3) {
   for (char4 = ';'; char4 <= 'z' ; ++char4) {

      ((char *)M)[0] = char0;
      ((char *)M)[1] = char1;
      ((char *)M)[2] = char2;
      ((char *)M)[3] = char3;
      ((char *)M)[4] = char4;

      //Initialize variables:
      uint32_t a0 = 0x67452301; 
      uint32_t b0 = 0xefcdab89;
      uint32_t c0 = 0x98badcfe;
      uint32_t d0 = 0x10325476;


       //Initialize hash value for this chunk:
       uint32_t A = a0;
       uint32_t B = b0;
       uint32_t C = c0;
       uint32_t D = d0;

       //Main loop:                                                 
       uint32_t F; 

       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xd76aa478 + M[0];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 7);

       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xe8c7b756 + M[1];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 12);

       F = (B & C) | (( ~ B) & D);
       F = F + A + 0x242070db + M[2];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 17);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xc1bdceee + M[3];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 22);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xf57c0faf + M[4];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 7);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0x4787c62a + M[6];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 12);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xa8304613 + M[6];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 17);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xfd469501 + M[7];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 22);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0x698098d8 + M[8];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 7);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0x8b44f7af + M[9];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 12);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xffff5bb1  + M[10];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 17);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0x895cd7be + M[11];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 22);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0x6b901122 + M[12];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 7);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xfd987193 + M[13];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 12);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0xa679438e + M[14];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 17);
 
       F = (B & C) | (( ~ B) & D);
       F = F + A + 0x49b40821 + M[15];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 22);
 
// ********************************************************************************************

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xf61e2562 + M[1];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 5);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xc040b340+ M[6];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 9);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0x265e5a51 + M[11];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 14);

       F = (D & B) | (( ~ D) & C);
       F = F + A +0xe9b6c7aa  + M[0];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 20);
 
       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xd62f105d + M[5];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 5);
 
       F = (D & B) | (( ~ D) & C);
       F = F + A + 0x2441453 + M[10];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 9);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xd8a1e681 + M[15];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 14);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xe7d3fbc8 + M[4];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 20);
 
       F = (D & B) | (( ~ D) & C);
       F = F + A + 0x21e1cde6 + M[9];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 5);
 
       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xc33707d6 + M[14];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 9);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xf4d50d87 + M[3];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 14);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0x455a14ed + M[8];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 20);
 
       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xa9e3e905 + M[13];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 5);
 
       F = (D & B) | (( ~ D) & C);
       F = F + A + 0xfcefa3f8 + M[2];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 9);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0x676f02d9 + M[7];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 14);

       F = (D & B) | (( ~ D) & C);
       F = F + A + 0x8d2a4c8a + M[12];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 20);
 
// ***************************************************************************************************

       F = B ^ C ^ D;
       F = F + A + 0xfffa3942 + M[5];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 4);
 
       F = B ^ C ^ D;
       F = F + A + 0x8771f681+ M[8];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 11);
 
       F = B ^ C ^ D;
       F = F + A +  0x6d9d6122+ M[11];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 16);
 
       F = B ^ C ^ D;
       F = F + A + 0xfde5380c + M[14];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 23);
 
       F = B ^ C ^ D;
       F = F + A + 0xa4beea44 + M[1];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 4);
 
       F = B ^ C ^ D;
       F = F + A + 0x4bdecfa9 + M[4];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 11);
 
       F = B ^ C ^ D;
       F = F + A +0xf6bb4b60  + M[7];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 16);
 
       F = B ^ C ^ D;
       F = F + A + 0xbebfbc70 + M[10];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 23);
 
       F = B ^ C ^ D;
       F = F + A + 0x289b7ec6 + M[13];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 4);
 
       F = B ^ C ^ D;
       F = F + A + 0xeaa127fa + M[0];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 11);
 
       F = B ^ C ^ D;
       F = F + A +  0xd4ef3085+ M[3];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 16);
 
       F = B ^ C ^ D;
       F = F + A + 0x4881d05 + M[6];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 23);
 
       F = B ^ C ^ D;
       F = F + A + 0xd9d4d039 + M[9];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 4);
 
       F = B ^ C ^ D;
       F = F + A + 0xe6db99e5 + M[12];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 11);
 
       F = B ^ C ^ D;
       F = F + A + 0x1fa27cf8 + M[15];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 16);
 
       F = B ^ C ^ D;
       F = F + A + 0xc4ac5665 + M[2];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 23);

// **********************************************************************************************

       F = C ^ (B | (~  D));
       F = F + A + 0xf4292244 + M[0];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 6);
    
       F = C ^ (B | (~  D));
       F = F + A + 0x432aff97 + M[7];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 10);
    
       F = C ^ (B | (~  D));
       F = F + A + 0xab9423a7 + M[14];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 15);
    
       F = C ^ (B | (~  D));
       F = F + A + 0xfc93a039 + M[5];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 21);
 
       F = C ^ (B | (~  D));
       F = F + A + 0x655b59c3 + M[12];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 6);
    
       F = C ^ (B | (~  D));
       F = F + A + 0x8f0ccc92 + M[3];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 10);
    
       F = C ^ (B | (~  D));
       F = F + A + 0xffeff47d + M[10];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 15);
    
       F = C ^ (B | (~  D));
       F = F + A + 0x85845dd1 + M[1];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 21);
 

       F = C ^ (B | (~  D));
       F = F + A + 0x6fa87e4f + M[8];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 6);
    
       F = C ^ (B | (~  D));
       F = F + A + 0xfe2ce6e0 + M[15];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 10);
    
       F = C ^ (B | (~  D));
       F = F + A + 0xa3014314 + M[6];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 15);
    
       F = C ^ (B | (~  D));
       F = F + A + 0x4e0811a1 + M[13];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 21);
 

       F = C ^ (B | (~  D));
       F = F + A + 0xf7537e82 + M[4];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 6);
    
       F = C ^ (B | (~  D));
       F = F + A + 0xbd3af235 + M[11];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 10);
    
       F = C ^ (B | (~  D));
       F = F + A + 0x2ad7d2bb + M[2];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 15);
    
       F = C ^ (B | (~  D));
       F = F + A + 0xeb86d391 + M[9];  
       A = D;
       D = C;
       C = B;
       B = B + LEFTROTATE(F, 21);

       //Add this chunk's hash to result so far:
       a0 = a0 + A;
       b0 = b0 + B;
       c0 = c0 + C;
       d0 = d0 + D;
  
       if ( targetA == a0
         && targetB == b0
         && targetC == c0
         && targetD == d0) {
            msg[0] = char0;
            msg[1] = char1;
            msg[2] = char2;
            msg[3] = char3;
            msg[4] = char4;
//            printf("hashcount : %lu\n", hashcount);
            return ;
         }
   } 
   }
   }
   }
   

}

int main( int argc, char * argv[] ) {

    // check arguments
    if (argc != 2) return 1;

    // allocate and prepare hash
    char hash[]=  "00000000000000000000000000000000";
    memcpy(hash,argv[1],32);
    uint32_t hostA;
    uint32_t hostB;
    uint32_t hostC;
    uint32_t hostD;

    loadHash(hash, &hostA, &hostB, &hostC, &hostD);


    char msg[9] = {0,0,0,0,0,0,0,0,0};

    char *dev_msg;


    uint32_t M[BUFFER_LEN];
    uint32_t * dev_M;
    pad("01234", M);


    hipMalloc( (void**)&dev_M , BUFFER_LEN ) ;
    hipMalloc( (void**)&dev_msg , 9 ) ;

    hipMemcpy( (void*) dev_M, (void*) M, BUFFER_LEN ,hipMemcpyHostToDevice ) ;

    find<<<THREADS,1>>>( hostA, hostB, hostC, hostD, dev_M, dev_msg );

    hipMemcpy( msg, dev_msg, 8,hipMemcpyDeviceToHost ) ;
    if (msg[0] == 0) {
         printf("key not found \n");
    } else {
      printf("<<< FOUND KEY : %s >>>\n", msg);
    }

    hipFree( dev_msg ) ;
    return 0;
}
